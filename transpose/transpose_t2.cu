#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

/*
Try 2
- read : coalesced
- write : uncoalesced
*/
__global__ transposeKernel_2D_0(float *A_d, float *B_d, long long n)
{

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < n && col < n)
    {
        B_d[col * n + row] = A_d[row * n + col];
    }
}

__global__ transposeKernel_2D_1(float *A_d, float *B_d, long long n) {

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < n && col < n)
    {
        B_d[row * n + col] = A_d[col * n + row];
    }
}