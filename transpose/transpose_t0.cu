#include <hip/hip_runtime.h>


/*
Try 0
- read : coalesced
- write : not coalesced
*/
__global__ void tranposeKernel_1D_0(float *A_d, float *B_d, long long n)
{

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int row = tid / n;
    int col = tid % n;

    if (row < n && col < n)
    {
        B_d[col * n + row] = A_d[row * n + col];
    }
}